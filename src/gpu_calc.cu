#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hipblas.h"
#include "gpu_util.h"
#include "gpu_calc.h"

#define CUDA_NAN __int_as_float(0x7fffffff)

#define checkcuerr() \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error %s at %s:%d\n", \
                hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


#define checkcublaserr(stat) \
    do { \
        if (stat != HIPBLAS_STATUS_SUCCESS) { \
            fprintf (stderr, "CUBLAS failed: %s:%d error: %s\n", __FILE__, __LINE__, _cudaGetErrorEnum(stat)); \
            exit(1); \
        } \
    } while (0)


static const char *_cudaGetErrorEnum(hipblasStatus_t error)
{
    switch (error)
    {
        case HIPBLAS_STATUS_SUCCESS:
            return "HIPBLAS_STATUS_SUCCESS";

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "HIPBLAS_STATUS_NOT_INITIALIZED";

        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "HIPBLAS_STATUS_ALLOC_FAILED";

        case HIPBLAS_STATUS_INVALID_VALUE:
            return "HIPBLAS_STATUS_INVALID_VALUE";

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "HIPBLAS_STATUS_ARCH_MISMATCH";

        case HIPBLAS_STATUS_NOT_SUPPORTED:
            return "HIPBLAS_STATUS_NOT_SUPPORTED";

        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "HIPBLAS_STATUS_MAPPING_ERROR";

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "HIPBLAS_STATUS_EXECUTION_FAILED";

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "HIPBLAS_STATUS_INTERNAL_ERROR";
    }

    return "<unknown>";
}



__global__ void
k_calc_value(
        float* values, 
        const float* gainsums, 
        const float* weightsums, 
        const float* board_compatibility, 
        const float pot, 
        const float cost, 
        const float exp)
{
    int hand_index = blockDim.x*blockIdx.x + threadIdx.x; 
    if (hand_index < 1326) {
        float weightsum = weightsums[hand_index];
        float equity;
        if (board_compatibility[hand_index] == 0.0f) {
            values[hand_index] = CUDA_NAN;
        } else {
            if (weightsum) {
                equity = gainsums[hand_index]/weightsum;
                values[hand_index] = powf(equity, exp) * pot - cost;
            } else {
                values[hand_index] = 0.0f;
            }
        }
    }
}


namespace riverev::gpucalc {

void
calc_node_values(
        const float* hand_compatibility, // C
        const float* board_compatibility, // b
        const float* ranking, // R
        const float* opponent_weights, // W
        const float pot,
        const float cost,
        const float exp,
        float* values,
        float* gainsums,
        float* weightsums)
{
    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipblasStatus_t stat;
    //hipblasGemmAlgo_t algo = CUBLAS_GEMM_ALGO1;
    hipblasGemmAlgo_t algo = HIPBLAS_GEMM_DEFAULT;
    
    int device;
    hipGetDevice(&device);
    hipblasHandle_t handle = reinterpret_cast<hipblasHandle_t>(util::gpu::get_device_handle(device));
        
    int m = 1326, n = 1, k = 1326;
    const void* L;
    const void* R;
    void* P;

    // R W = A (gainsums)
    L = (const void*)(ranking);
    R = (const void*)(opponent_weights);
    P = (void*)(gainsums);
    stat = hipblasGemmEx(
        handle, 
        HIPBLAS_OP_T, HIPBLAS_OP_N, 
        m, n, k, 
        &alpha, 
        L, HIP_R_32F, m, 
        R, HIP_R_32F, k, 
        &beta, 
        P, HIP_R_32F, m,
        HIPBLAS_COMPUTE_32F, algo
    );
    checkcublaserr(stat);
    hipDeviceSynchronize();
    
    // C W = B (weightsums)
    L = (const void*)(hand_compatibility);
    R = (const void*)(opponent_weights);
    P = (void*)(weightsums);
    stat = hipblasGemmEx(
        handle, 
        HIPBLAS_OP_T, HIPBLAS_OP_N, 
        m, n, k, 
        &alpha, 
        L, HIP_R_32F, m, 
        R, HIP_R_32F, k, 
        &beta, 
        P, HIP_R_32F, m,
        HIPBLAS_COMPUTE_32F, algo
    );
    checkcublaserr(stat);
    hipDeviceSynchronize();
    
    const int blockDimX = 256;
    dim3 dimBlock(blockDimX);
    dim3 dimGrid((1326+blockDimX-1)/blockDimX);

    k_calc_value<<<dimGrid, dimBlock>>>(
            values, 
            gainsums, 
            weightsums, 
            board_compatibility, 
            pot, 
            cost, 
            exp);
    hipDeviceSynchronize();
    checkcuerr();
}


} // riverev::gpucalc
