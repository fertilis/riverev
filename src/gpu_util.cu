#include <stdio.h>
#include <stdint.h>
#include "hip/hip_runtime.h"
#include ""
#include "hipblas.h"
#include "gpu_util.h"

//#define IDX2C(i,j,ld) (((j)*(ld))+(i))

#define ALIGNED(x,y) (((x)%(y)) ? (x)+(y)-((x)%(y)) : x)
#define checkcuerr() \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error %s at %s:%d\n", \
                hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


#define checkcublaserr(stat) \
    do { \
        if (stat != HIPBLAS_STATUS_SUCCESS) { \
            fprintf (stderr, "CUBLAS failed: %s:%d\n", __FILE__, __LINE__); \
            exit(1); \
        } \
    } while (0)



#define MAX(a,b) (((a)>(b))?(a):(b))
#define MIN(a,b) (((a)<(b))?(a):(b))
#define CLIP(a,b,c) (((a)<(b))?(b):((a)>(c)?(c):(a)))


namespace util::gpu {

//hipblasHandle_t HANDLE;
hipblasHandle_t HANDLES[2] = {0};


size_t get_device_handle(int device) {
    return reinterpret_cast<size_t>(HANDLES[device]);
}


void cu_cublasCreate_once(int device)
{
    if (device < 2 && !HANDLES[device]) {
        int cur_device;
        hipGetDevice(&cur_device);
        hipblasStatus_t stat;
        hipSetDevice(device);
        stat = hipblasCreate(&HANDLES[device]);
        checkcublaserr(stat);
        hipSetDevice(cur_device);
    }
//    hipblasStatus_t stat = hipblasCreate(&HANDLE);
//    checkcublaserr(stat);
}


void cu_cudaDeviceSynchronize()
{
    hipDeviceSynchronize();
    checkcuerr();
}


void cu_cudaFree(void* ptr)
{
    hipFree(ptr);
    checkcuerr();
}


void cu_cudaFreeHost(void* ptr)
{
    hipHostFree(ptr);
    checkcuerr();
}

void cu_cudaMallocHost(void** ptr, unsigned size)
{
    hipHostMalloc(ptr, size);
    checkcuerr();
}

void cu_cudaMalloc(void** ptr, unsigned size)
{
    hipMalloc(ptr, size);
    checkcuerr();
    hipDeviceSynchronize();
    checkcuerr();
}


void cu_cudaMemcpy_HD(void* dst, const void* src, size_t count)
{
    hipMemcpy(dst, src, count, hipMemcpyHostToDevice);
    checkcuerr();
}

void cu_cudaMemcpy_DH(void* dst, const void* src, size_t count)
{
    hipMemcpy(dst, src, count, hipMemcpyDeviceToHost);
    checkcuerr();
}

void cu_cudaMemcpy_DD(void* dst, const void* src, size_t count)
{
    hipMemcpy(dst, src, count, hipMemcpyDeviceToDevice);
    checkcuerr();
}

void cu_cudaMemGetInfo(size_t* free, size_t* total)
{
    hipMemGetInfo(free, total);
}

void cu_cudaSetDevice(int device)
{
    hipSetDevice(device);
    hipDeviceSynchronize();
    checkcuerr();
}

int cu_cudaGetDevice()
{
    int device;
    hipGetDevice(&device);
    return device;
}

} // util::gpu
